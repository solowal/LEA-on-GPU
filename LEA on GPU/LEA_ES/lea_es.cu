#include "hip/hip_runtime.h"
#include "lea_es.cuh"
#include <stdio.h>

/// @brief C version
void LEA128_Keyschedule(u32 pdRndKeys[LEA128_NUM_RNDS * LEA128_RNDKEY_WORD_LEN],
						const u32 pbKey[LEA128_KEY_WORD_LEN])
{
	u32 delta[4] = {0xc3efe9db, 0x44626b02, 0x79e27c8a, 0x78df30ec};
	u32 T[4] = {
		0x0,
	};

	T[0] = pbKey[0];
	T[1] = pbKey[1];
	T[2] = pbKey[2];
	T[3] = pbKey[3];

	for (int i = 0; i < LEA128_NUM_RNDS; i++)
	{
		T[0] = ROL(T[0] + ROL(delta[i & 3], i), 1);
		T[1] = ROL(T[1] + ROL(delta[i & 3], i + 1), 3);
		T[2] = ROL(T[2] + ROL(delta[i & 3], i + 2), 6);
		T[3] = ROL(T[3] + ROL(delta[i & 3], i + 3), 11);

		pdRndKeys[i * LEA128_RNDKEY_WORD_LEN + 0] = T[0];
		pdRndKeys[i * LEA128_RNDKEY_WORD_LEN + 1] = T[1];
		pdRndKeys[i * LEA128_RNDKEY_WORD_LEN + 2] = T[2];
		pdRndKeys[i * LEA128_RNDKEY_WORD_LEN + 3] = T[3];
		// pdRndKeys[i][3] = T[1];
		// pdRndKeys[i][4] = T[3];
		// pdRndKeys[i][5] = T[1];
	}
}

void LEA192_Keyschedule(u32 pdRndKeys[LEA192_NUM_RNDS * LEA_RNDKEY_WORD_LEN],
						const u32 pbKey[LEA192_KEY_WORD_LEN])
{
	u32 delta[6] = {0xc3efe9db, 0x44626b02, 0x79e27c8a, 0x78df30ec, 0x715ea49e, 0xc785da0a};
	u32 T[6] = {
		0x0,
	};

	T[0] = pbKey[0];
	T[1] = pbKey[1];
	T[2] = pbKey[2];
	T[3] = pbKey[3];
	T[4] = pbKey[4];
	T[5] = pbKey[5];

	for (int i = 0; i < LEA192_NUM_RNDS; i++)
	{
		T[0] = ROL(T[0] + ROL(delta[i % 6], i + 0), 1);
		T[1] = ROL(T[1] + ROL(delta[i % 6], i + 1), 3);
		T[2] = ROL(T[2] + ROL(delta[i % 6], i + 2), 6);
		T[3] = ROL(T[3] + ROL(delta[i % 6], i + 3), 11);
		T[4] = ROL(T[4] + ROL(delta[i % 6], i + 4), 13);
		T[5] = ROL(T[5] + ROL(delta[i % 6], i + 5), 17);

		pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 0] = T[0];
		pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 1] = T[1];
		pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 2] = T[2];
		pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 3] = T[3];
		pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 4] = T[4];
		pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 5] = T[5];
	}
}

void LEA256_Keyschedule(u32 pdRndKeys[LEA256_NUM_RNDS * LEA_RNDKEY_WORD_LEN],
						const u32 pbKey[LEA256_KEY_WORD_LEN])
{
	u32 delta[8] = {0xc3efe9db, 0x44626b02, 0x79e27c8a, 0x78df30ec, 0x715ea49e, 0xc785da0a, 0xe04ef22a, 0xe5c40957};
	u32 T[8] = {
		0x0,
	};

	T[0] = pbKey[0];
	T[1] = pbKey[1];
	T[2] = pbKey[2];
	T[3] = pbKey[3];
	T[4] = pbKey[4];
	T[5] = pbKey[5];
	T[6] = pbKey[6];
	T[7] = pbKey[7];

	for (int i = 0; i < LEA256_NUM_RNDS; i++)
	{
		T[(6 * i) % 8] = ROL(T[(6 * i) % 8] + ROL(delta[i % 8], i), 1);
		T[(6 * i + 1) % 8] = ROL(T[(6 * i + 1) % 8] + ROL(delta[i % 8], i + 1), 3);
		T[(6 * i + 2) % 8] = ROL(T[(6 * i + 2) % 8] + ROL(delta[i % 8], i + 2), 6);
		T[(6 * i + 3) % 8] = ROL(T[(6 * i + 3) % 8] + ROL(delta[i % 8], i + 3), 11);
		T[(6 * i + 4) % 8] = ROL(T[(6 * i + 4) % 8] + ROL(delta[i % 8], i + 4), 13);
		T[(6 * i + 5) % 8] = ROL(T[(6 * i + 5) % 8] + ROL(delta[i % 8], i + 5), 17);

		pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 0] = T[(6 * i) % 8];
		pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 1] = T[(6 * i + 1) % 8];
		pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 2] = T[(6 * i + 2) % 8];
		pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 3] = T[(6 * i + 3) % 8];
		pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 4] = T[(6 * i + 4) % 8];
		pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 5] = T[(6 * i + 5) % 8];
	}
}

void LEA128_EncryptBlk(u32 pbDst[LEA_BLK_WORD_LEN],
					   const u32 pbSrc[LEA_BLK_WORD_LEN],
					   const u32 pdRndKeys[LEA128_NUM_RNDS * LEA128_RNDKEY_WORD_LEN])
{
	u32 X0, X1, X2, X3;
	u32 temp;

	X0 = pbSrc[0];
	X1 = pbSrc[1];
	X2 = pbSrc[2];
	X3 = pbSrc[3];

	for (int i = 0; i < LEA128_NUM_RNDS; i++)
	{
		X3 = ROR((X2 ^ pdRndKeys[i * LEA128_RNDKEY_WORD_LEN + 3]) + (X3 ^ pdRndKeys[i * LEA128_RNDKEY_WORD_LEN + 1]), 3);
		X2 = ROR((X1 ^ pdRndKeys[i * LEA128_RNDKEY_WORD_LEN + 2]) + (X2 ^ pdRndKeys[i * LEA128_RNDKEY_WORD_LEN + 1]), 5);
		X1 = ROL((X0 ^ pdRndKeys[i * LEA128_RNDKEY_WORD_LEN + 0]) + (X1 ^ pdRndKeys[i * LEA128_RNDKEY_WORD_LEN + 1]), 9);
		temp = X0;
		X0 = X1;
		X1 = X2;
		X2 = X3;
		X3 = temp;
	}

	pbDst[0] = X0;
	pbDst[1] = X1;
	pbDst[2] = X2;
	pbDst[3] = X3;
}

void LEA192_EncryptBlk(u32 pbDst[LEA_BLK_WORD_LEN],
					   const u32 pbSrc[LEA_BLK_WORD_LEN],
					   const u32 pdRndKeys[LEA192_NUM_RNDS * LEA_RNDKEY_WORD_LEN])
{
	u32 X0, X1, X2, X3;
	u32 temp;

	X0 = pbSrc[0];
	X1 = pbSrc[1];
	X2 = pbSrc[2];
	X3 = pbSrc[3];

	for (int i = 0; i < LEA192_NUM_RNDS; i++)
	{
		X3 = ROR((X2 ^ pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 4]) + (X3 ^ pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 5]), 3);
		X2 = ROR((X1 ^ pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 2]) + (X2 ^ pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 3]), 5);
		X1 = ROL((X0 ^ pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 0]) + (X1 ^ pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 1]), 9);
		temp = X0;
		X0 = X1;
		X1 = X2;
		X2 = X3;
		X3 = temp;
	}

	pbDst[0] = X0;
	pbDst[1] = X1;
	pbDst[2] = X2;
	pbDst[3] = X3;
}

void LEA256_EncryptBlk(u32 pbDst[LEA_BLK_WORD_LEN],
					   const u32 pbSrc[LEA_BLK_WORD_LEN],
					   const u32 pdRndKeys[LEA256_NUM_RNDS * LEA_RNDKEY_WORD_LEN])
{
	u32 X0, X1, X2, X3;
	u32 temp;

	X0 = pbSrc[0];
	X1 = pbSrc[1];
	X2 = pbSrc[2];
	X3 = pbSrc[3];

	for (int i = 0; i < LEA256_NUM_RNDS; i++)
	{
		X3 = ROR((X2 ^ pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 4]) + (X3 ^ pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 5]), 3);
		X2 = ROR((X1 ^ pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 2]) + (X2 ^ pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 3]), 5);
		X1 = ROL((X0 ^ pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 0]) + (X1 ^ pdRndKeys[i * LEA_RNDKEY_WORD_LEN + 1]), 9);
		temp = X0;
		X0 = X1;
		X1 = X2;
		X2 = X3;
		X3 = temp;
	}

	pbDst[0] = X0;
	pbDst[1] = X1;
	pbDst[2] = X2;
	pbDst[3] = X3;
}

/////////////////////////////////////////

__device__ __forceinline__
	u32
	ROR_device(u32 input, u32 shift)
{
	u32 r;
	asm("{                  \n\t"
		"shf.r.wrap.b32 %0, %1, %2, %3; \n\t"
		"}"
		: "=r"(r)							   // 출력: 가상 레지스터 할당
		: "r"(input), "r"(input), "r"(shift)); // 입력: 가상 레지스터 할당
	return r;
}

__device__ __forceinline__
	u32
	ROL_device(u32 input, u32 shift)
{
	u32 r;
	asm("{                  \n\t"
		"shf.l.wrap.b32 %0, %1, %2, %3; \n\t"
		"}"
		: "=r"(r)							   // 출력: 가상 레지스터 할당
		: "r"(input), "r"(input), "r"(shift)); // 입력: 가상 레지스터 할당
	return r;
}

__global__ void LEA_128_ES_SharedMemory_TEST(u32 *u_plain, u32 *u_masterkey, u32 *u_delta, u64 *range)
{
	u64 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ u32 dtS[LEA_DELTA_WORD_LEN];
	if (threadIdx.x < LEA_DELTA_WORD_LEN)
	{
		dtS[threadIdx.x] = u_delta[threadIdx.x];
	}
	__syncthreads();

	u32 X0, X1, X2, X3;
	u32 T[4];
	u32 temp;

	X0 = u_plain[0];
	X1 = u_plain[1];
	X2 = u_plain[2];
	X3 = u_plain[3];

	if (threadIndex == 0)
	{
		printf("threadIndex : %llu\n", threadIndex);
		printf("Plaintext   : %08X %08X %08X %08X\n", X0, X1, X2, X3);
		printf("-------------------------------\n");
	}

	T[0] = u_masterkey[0];
	T[1] = u_masterkey[1];
	T[2] = u_masterkey[2];
	T[3] = u_masterkey[3];

	for (int i = 0; i < LEA128_NUM_RNDS; i++)
	{
		// key gen
		T[0] = ROL_device(T[0] + ROL_device(dtS[i & 3], i), 1);
		T[1] = ROL_device(T[1] + ROL_device(dtS[i & 3], i + 1), 3);
		T[2] = ROL_device(T[2] + ROL_device(dtS[i & 3], i + 2), 6);
		T[3] = ROL_device(T[3] + ROL_device(dtS[i & 3], i + 3), 11);

		// encryption
		X3 = ROR_device((X2 ^ T[3]) + (X3 ^ T[1]), 3);
		X2 = ROR_device((X1 ^ T[2]) + (X2 ^ T[1]), 5);
		X1 = ROL_device((X0 ^ T[0]) + (X1 ^ T[1]), 9);
		temp = X0;
		X0 = X1;
		X1 = X2;
		X2 = X3;
		X3 = temp;
		if (threadIndex == 0)
		{
			printf("Ciphertext  %d : %08X %08X %08X %08X\n", i, X0, X1, X2, X3);
		}
	}

	if (threadIndex == 0)
	{
		printf("threadIndex : %llu\n", threadIndex);
		printf("Ciphertext   : %08X %08X %08X %08X\n", X0, X1, X2, X3);
		printf("-------------------------------\n");
	}
}

__global__ void LEA_128_ES_SharedMemory(u32 *u_plain, u32 *u_cipher, u32 *u_masterkey, u32 *u_delta, u64 *range)
{
	u64 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ u32 dtS[LEA_DELTA_WORD_LEN];
	__shared__ u32 ctS[LEA_BLK_WORD_LEN];

	if (threadIdx.x < LEA_DELTA_WORD_LEN)
	{
		dtS[threadIdx.x] = u_delta[threadIdx.x];
		ctS[threadIdx.x] = u_cipher[threadIdx.x];
	}
	__syncthreads();

	u32 X0, X1, X2, X3;
	u32 T[4];
	u32 temp;

	// X0 = u_plain[0];
	// X1 = u_plain[1];
	// X2 = u_plain[2];
	// X3 = u_plain[3];

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = u_masterkey[0];
	rk1Init = u_masterkey[1];
	rk2Init = u_masterkey[2];
	rk3Init = u_masterkey[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = u_plain[0];
	pt1Init = u_plain[1];
	pt2Init = u_plain[2];
	pt3Init = u_plain[3];

	u64 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	// if (threadIndex == 0) {
	// 	printf("threadIndex : %llu\n", threadIndex);
	// 	printf("Plaintext   : %08X %08X %08X %08X\n", X0, X1, X2, X3);
	// 	printf("-------------------------------\n");
	// }

	// T[0] = u_masterkey[0];
	// T[1] = u_masterkey[1];
	// T[2] = u_masterkey[2];
	// T[3] = u_masterkey[3];
	for (u64 rangeCount = 0; rangeCount < threadRange; rangeCount++)
	{
		// u32 rk0, rk1, rk2, rk3;
		T[0] = rk0Init;
		T[1] = rk1Init;
		T[2] = rk2Init;
		T[3] = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		// u32 s0, s1, s2, s3;
		X0 = pt0Init;
		X1 = pt1Init;
		X2 = pt2Init;
		X3 = pt3Init;

		for (int i = 0; i < LEA128_NUM_RNDS; i++)
		{
			// key gen
			T[0] = ROL_device(T[0] + ROL_device(dtS[i & 3], i), 1);
			T[1] = ROL_device(T[1] + ROL_device(dtS[i & 3], i + 1), 3);
			T[2] = ROL_device(T[2] + ROL_device(dtS[i & 3], i + 2), 6);
			T[3] = ROL_device(T[3] + ROL_device(dtS[i & 3], i + 3), 11);

			// encryption
			X3 = ROR_device((X2 ^ T[3]) + (X3 ^ T[1]), 3);
			X2 = ROR_device((X1 ^ T[2]) + (X2 ^ T[1]), 5);
			X1 = ROL_device((X0 ^ T[0]) + (X1 ^ T[1]), 9);
			temp = X0;
			X0 = X1;
			X1 = X2;
			X2 = X3;
			X3 = temp;
			// if (threadIndex == 0) {
			// 	printf("Ciphertext  %d : %08X %08X %08X %08X\n", i, X0, X1, X2, X3);
			// }
		}

		if (X0 == ctS[0])
		{
			if (X1 == ctS[1])
			{
				if (X2 == ctS[2])
				{
					if (X3 == ctS[3])
					{
						printf("threadIndex : %llu\n", threadIndex);
						printf("threadRange : %llu\n", threadRange);
						printf("Ciphertext   : %08X %08X %08X %08X\n", X0, X1, X2, X3);
						printf("-------------------------------\n");
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32)
		{
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
	// if (threadIndex == 0) {
	// 	printf("threadIndex : %llu\n", threadIndex);
	// 	printf("Ciphertext   : %08X %08X %08X %08X\n", X0, X1, X2, X3);
	// 	printf("-------------------------------\n");
	// }
}

void LEA_128_ES_ShaerdMemory_main()
{
	// master key
	// 0x3C2D1E0F, 0x78695A4B, 0xB4A59687, 0xF0E1D2C3
	// plaintext
	// 0x13121110, 0x17161514, 0x1B1A1918, 0x1F1E1D1C
	// ciphertext
	// 0x354EC89F, 0x18C6C628, 0xA7C73255, 0xFD8B6404

	u32 pdRndKeys[LEA128_NUM_RNDS * LEA128_RNDKEY_WORD_LEN];
	u32 masterKeys[LEA128_KEY_WORD_LEN] = {0x3C2D1E0F, 0x78695A4B, 0xB4A59687, 0xF0E1D2C3};
	u32 plaintext[LEA_BLK_WORD_LEN] = {0x13121110, 0x17161514, 0x1B1A1918, 0x1F1E1D1C};
	u32 delta[LEA_DELTA_WORD_LEN] = {0xc3efe9db, 0x44626b02, 0x79e27c8a, 0x78df30ec};
	u32 ciphertext[LEA_BLK_WORD_LEN] = {
		0,
	};

	// checkDeviceProperties();
	// C test
	LEA128_Keyschedule(pdRndKeys, masterKeys);
	LEA128_EncryptBlk(ciphertext, plaintext, pdRndKeys);
	printf("LEA128 ciphertext: 0x%08X, 0x%08X, 0x%08X, 0x%08X\n", ciphertext[0], ciphertext[1], ciphertext[2], ciphertext[3]);

	// LEA192_Keyschedule(pdRndKeys,masterKeys);
	// LEA192_EncryptBlk(ciphertext,plaintext,pdRndKeys);
	// printf("LEA192 ciphertext: 0x%08X, 0x%08X, 0x%08X, 0x%08X\n", ciphertext[0], ciphertext[1], ciphertext[2], ciphertext[3]);

	// LEA256_Keyschedule(pdRndKeys,masterKeys);
	// LEA256_EncryptBlk(ciphertext,plaintext,pdRndKeys);
	// printf("LEA256 ciphertext: 0x%08X, 0x%08X, 0x%08X, 0x%08X\n", ciphertext[0], ciphertext[1], ciphertext[2], ciphertext[3]);

	// 데이터 크기:     (temporal) 16 bytes * 256 units = 4096 bytes
	// round key 크기:             16 bytes * 24 rounds= 384 bytes

	// size_t size = INPUT_DATA_BYTE_LEN;

	u32 *u_plain, *u_masterkey, *u_delta, *u_cipher;

	gpuErrorCheck(hipMallocManaged(&u_plain, LEA_BLK_WORD_LEN * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&u_masterkey, LEA128_KEY_WORD_LEN * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&u_delta, LEA_DELTA_WORD_LEN * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&u_cipher, LEA_BLK_WORD_LEN * sizeof(u32)));

	for (int i = 0; i < LEA_BLK_WORD_LEN; i++)
	{
		u_plain[i] = plaintext[i];
	}
	for (int i = 0; i < (LEA128_KEY_WORD_LEN); i++)
	{
		u_masterkey[i] = masterKeys[i];
	}
	for (int i = 0; i < (LEA_DELTA_WORD_LEN); i++)
	{
		u_delta[i] = delta[i];
	}
	for (int i = 0; i < LEA_BLK_WORD_LEN; i++)
	{
		u_cipher[i] = ciphertext[i];
	}

	printf("-------------------------------\n");
	u64 *range = calculateRange();

	clock_t beginTime = clock();
	// LEA_128_CTR_SharedMemory_TEST<<<BLOCKS,THREADS>>>(u_plain, u_roundkey, range);
	// LEA_128_CTR_SharedMemory<<<BLOCKS,THREADS>>>(u_plain, u_roundkey, range);
	LEA_128_ES_SharedMemory_TEST<<<4, 4>>>(u_plain, u_masterkey, u_delta, range);
	// LEA_128_ES_SharedMemory<<<BLOCKS, THREADS>>>(u_plain, u_cipher, u_masterkey, u_delta, range);
	gpuErrorCheck(hipDeviceSynchronize());
	printf("Time elapsed: %f sec\n", float(clock() - beginTime) / CLOCKS_PER_SEC);
	printf("-------------------------------\n");
	printLastCUDAError();

	hipFree(u_plain);
	hipFree(u_masterkey);
	hipFree(u_delta);
}

__global__ void LEA_192_ES_SharedMemory(u32 *u_plain, u32 *u_cipher, u32 *u_masterkey, u32 *u_delta, u64 *range)
{
	u64 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ u32 dtS[LEA_DELTA_WORD_LEN];
	__shared__ u32 ctS[LEA_BLK_WORD_LEN];

	if (threadIdx.x < LEA_DELTA_WORD_LEN)
	{
		if (threadIdx.x < LEA_BLK_WORD_LEN)
			ctS[threadIdx.x] = u_cipher[threadIdx.x];
		dtS[threadIdx.x] = u_delta[threadIdx.x];
	}
	__syncthreads();

	u32 X0, X1, X2, X3;
	u32 T[6];
	u32 temp;

	u32 rk0Init, rk1Init, rk2Init, rk3Init, rk4Init, rk5Init;
	rk0Init = u_masterkey[0];
	rk1Init = u_masterkey[1];
	rk2Init = u_masterkey[2];
	rk3Init = u_masterkey[3];
	rk4Init = u_masterkey[4];
	rk5Init = u_masterkey[5];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = u_plain[0];
	pt1Init = u_plain[1];
	pt2Init = u_plain[2];
	pt3Init = u_plain[3];

	u64 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	// rk2Init = rk2Init + threadRangeStart / MAX_U32;
	// rk3Init = rk3Init + threadRangeStart % MAX_U32;
	rk4Init = rk4Init + threadRangeStart / MAX_U32;
	rk5Init = rk5Init + threadRangeStart % MAX_U32;

	for (u64 rangeCount = 0; rangeCount < threadRange; rangeCount++)
	{
		// u32 rk0, rk1, rk2, rk3;
		T[0] = rk0Init;
		T[1] = rk1Init;
		T[2] = rk2Init;
		T[3] = rk3Init;
		T[4] = rk4Init;
		T[5] = rk5Init;

		// Create plaintext as 32 bit unsigned integers
		// u32 s0, s1, s2, s3;
		X0 = pt0Init;
		X1 = pt1Init;
		X2 = pt2Init;
		X3 = pt3Init;

		for (int i = 0; i < LEA192_NUM_RNDS; i++)
		{
			// key gen
			T[0] = ROL_device(T[0] + ROL_device(dtS[i % 6], i + 0), 1);
			T[1] = ROL_device(T[1] + ROL_device(dtS[i % 6], i + 1), 3);
			T[2] = ROL_device(T[2] + ROL_device(dtS[i % 6], i + 2), 6);
			T[3] = ROL_device(T[3] + ROL_device(dtS[i % 6], i + 3), 11);
			T[4] = ROL_device(T[4] + ROL_device(dtS[i % 6], i + 4), 13);
			T[5] = ROL_device(T[5] + ROL_device(dtS[i % 6], i + 5), 17);

			// encryption
			X3 = ROR_device((X2 ^ T[4]) + (X3 ^ T[5]), 3);
			X2 = ROR_device((X1 ^ T[2]) + (X2 ^ T[3]), 5);
			X1 = ROL_device((X0 ^ T[0]) + (X1 ^ T[1]), 9);
			temp = X0;
			X0 = X1;
			X1 = X2;
			X2 = X3;
			X3 = temp;
		}

		if (X0 == ctS[0])
		{
			if (X1 == ctS[1])
			{
				if (X2 == ctS[2])
				{
					if (X3 == ctS[3])
					{
						printf("threadIndex : %llu\n", threadIndex);
						printf("threadRange : %llu\n", threadRange);
						printf("Ciphertext   : %08X %08X %08X %08X\n", X0, X1, X2, X3);
						printf("-------------------------------\n");
					}
				}
			}
		}

		// Overflow
		if (rk5Init == MAX_U32)
		{
			rk4Init++;
		}
		// Create key as 32 bit unsigned integers
		rk5Init++;
	}
}

void LEA_192_ES_ShaerdMemory_main()
{
	// master key
	// 0x3C2D1E0F, 0x78695A4B, 0xB4A59687, 0xF0E1D2C3, 0xc3d2e1f0, 0x8796a5b4
	// plaintext
	// 0x23222120, 0x27262524, 0x2B2A2928, 0x2F2E2D2C
	// ciphertext
	// 0x325eb96f, 0x871bad5a, 0x35f5dc8c, 0xf2c67476

	u32 pdRndKeys[LEA192_NUM_RNDS * LEA_RNDKEY_WORD_LEN];
	u32 masterKeys[LEA192_KEY_WORD_LEN] = {0x3C2D1E0F, 0x78695A4B, 0xB4A59687, 0xF0E1D2C3, 0xc3d2e1f0, 0x8796a5b4};
	u32 plaintext[LEA_BLK_WORD_LEN] = {0x23222120, 0x27262524, 0x2B2A2928, 0x2F2E2D2C};
	u32 delta[LEA_DELTA_WORD_LEN] = {0xc3efe9db, 0x44626b02, 0x79e27c8a, 0x78df30ec, 0x715ea49e, 0xc785da0a, 0xe04ef22a, 0xe5c40957};
	u32 ciphertext[LEA_BLK_WORD_LEN] = {
		0,
	};

	LEA192_Keyschedule(pdRndKeys, masterKeys);
	LEA192_EncryptBlk(ciphertext, plaintext, pdRndKeys);
	printf("LEA192 ciphertext: 0x%08X, 0x%08X, 0x%08X, 0x%08X\n", ciphertext[0], ciphertext[1], ciphertext[2], ciphertext[3]);

	u32 *u_plain, *u_masterkey, *u_delta, *u_cipher;

	gpuErrorCheck(hipMallocManaged(&u_plain, LEA_BLK_WORD_LEN * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&u_masterkey, LEA192_KEY_WORD_LEN * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&u_delta, LEA_DELTA_WORD_LEN * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&u_cipher, LEA_BLK_WORD_LEN * sizeof(u32)));

	for (int i = 0; i < LEA_BLK_WORD_LEN; i++)
	{
		u_plain[i] = plaintext[i];
	}
	for (int i = 0; i < (LEA192_KEY_WORD_LEN); i++)
	{
		u_masterkey[i] = masterKeys[i];
	}
	for (int i = 0; i < (LEA_DELTA_WORD_LEN); i++)
	{
		u_delta[i] = delta[i];
	}
	for (int i = 0; i < LEA_BLK_WORD_LEN; i++)
	{
		u_cipher[i] = ciphertext[i];
	}

	printf("-------------------------------\n");
	u64 *range = calculateRange();

	clock_t beginTime = clock();
	LEA_192_ES_SharedMemory<<<BLOCKS, THREADS>>>(u_plain, u_cipher, u_masterkey, u_delta, range);
	gpuErrorCheck(hipDeviceSynchronize());
	printf("Time elapsed: %f sec\n", float(clock() - beginTime) / CLOCKS_PER_SEC);
	printf("-------------------------------\n");
	printLastCUDAError();

	hipFree(u_plain);
	hipFree(u_masterkey);
	hipFree(u_delta);
}


__global__ void LEA_256_ES_SharedMemory(u32 *u_plain, u32 *u_cipher, u32 *u_masterkey, u32 *u_delta, u64 *range)
{
	u64 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ u32 dtS[LEA_DELTA_WORD_LEN];
	__shared__ u32 ctS[LEA_BLK_WORD_LEN];

	if (threadIdx.x < LEA_DELTA_WORD_LEN)
	{
		if (threadIdx.x < LEA_BLK_WORD_LEN)
			ctS[threadIdx.x] = u_cipher[threadIdx.x];
		dtS[threadIdx.x] = u_delta[threadIdx.x];
	}
	__syncthreads();

	u32 X0, X1, X2, X3;
	u32 T[8];
	u32 temp;

	// X0 = u_plain[0];
	// X1 = u_plain[1];
	// X2 = u_plain[2];
	// X3 = u_plain[3];

	u32 rk0Init, rk1Init, rk2Init, rk3Init, rk4Init, rk5Init, rk6Init, rk7Init;
	rk0Init = u_masterkey[0];
	rk1Init = u_masterkey[1];
	rk2Init = u_masterkey[2];
	rk3Init = u_masterkey[3];
	rk4Init = u_masterkey[4];
	rk5Init = u_masterkey[5];
	rk6Init = u_masterkey[6];
	rk7Init = u_masterkey[7];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = u_plain[0];
	pt1Init = u_plain[1];
	pt2Init = u_plain[2];
	pt3Init = u_plain[3];

	u64 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	// rk2Init = rk2Init + threadRangeStart / MAX_U32;
	// rk3Init = rk3Init + threadRangeStart % MAX_U32;
	rk6Init = rk6Init + threadRangeStart / MAX_U32;
	rk7Init = rk7Init + threadRangeStart % MAX_U32;

	for (u64 rangeCount = 0; rangeCount < threadRange; rangeCount++)
	{
		// u32 rk0, rk1, rk2, rk3;
		T[0] = rk0Init;
		T[1] = rk1Init;
		T[2] = rk2Init;
		T[3] = rk3Init;
		T[4] = rk4Init;
		T[5] = rk5Init;
		T[6] = rk6Init;
		T[7] = rk7Init;

		// Create plaintext as 32 bit unsigned integers
		// u32 s0, s1, s2, s3;
		X0 = pt0Init;
		X1 = pt1Init;
		X2 = pt2Init;
		X3 = pt3Init;

		for (int i = 0; i < LEA256_NUM_RNDS; i++)
		{
			// key gen
			T[(6 * i + 0) % 8] = ROL_device(T[(6 * i + 0) % 8] + ROL_device(dtS[i % 8], i + 0), 1);
			T[(6 * i + 1) % 8] = ROL_device(T[(6 * i + 1) % 8] + ROL_device(dtS[i % 8], i + 1), 3);
			T[(6 * i + 2) % 8] = ROL_device(T[(6 * i + 2) % 8] + ROL_device(dtS[i % 8], i + 2), 6);
			T[(6 * i + 3) % 8] = ROL_device(T[(6 * i + 3) % 8] + ROL_device(dtS[i % 8], i + 3), 11);
			T[(6 * i + 4) % 8] = ROL_device(T[(6 * i + 4) % 8] + ROL_device(dtS[i % 8], i + 4), 13);
			T[(6 * i + 5) % 8] = ROL_device(T[(6 * i + 5) % 8] + ROL_device(dtS[i % 8], i + 5), 17);

			// encryption
			X3 = ROR_device((X2 ^ T[(6 * i + 4) % 8]) + (X3 ^ T[(6 * i + 5) % 8]), 3);
			X2 = ROR_device((X1 ^ T[(6 * i + 2) % 8]) + (X2 ^ T[(6 * i + 3) % 8]), 5);
			X1 = ROL_device((X0 ^ T[(6 * i + 0) % 8]) + (X1 ^ T[(6 * i + 1) % 8]), 9);
			temp = X0;
			X0 = X1;
			X1 = X2;
			X2 = X3;
			X3 = temp;
		}

		if (X0 == ctS[0])
		{
			if (X1 == ctS[1])
			{
				if (X2 == ctS[2])
				{
					if (X3 == ctS[3])
					{
						printf("threadIndex : %llu\n", threadIndex);
						printf("threadRange : %llu\n", threadRange);
						printf("Ciphertext   : %08X %08X %08X %08X\n", X0, X1, X2, X3);
						printf("-------------------------------\n");
					}
				}
			}
		}

		// Overflow
		if (rk7Init == MAX_U32)
		{
			rk6Init++;
		}

		// Create key as 32 bit unsigned integers
		rk7Init++;
	}
}


void LEA_256_ES_ShaerdMemory_main()
{
	// master key
	// 0x3C2D1E0F, 0x78695A4B, 0xB4A59687, 0xF0E1D2C3, 0xc3d2e1f0, 0x8796a5b4, 0x4b5a6978, 0x0f1e2d3c
	// plaintext
	// 0x33323130, 0x37363534, 0x3B3A3938, 0x3F3E3D3C
	// ciphertext
	// 0xf6af51d6, 0xc189b147, 0xca00893a, 0x97e1f927

	u32 pdRndKeys[LEA256_NUM_RNDS * LEA_RNDKEY_WORD_LEN];
	u32 masterKeys[LEA256_KEY_WORD_LEN] = {0x3C2D1E0F, 0x78695A4B, 0xB4A59687, 0xF0E1D2C3, 0xc3d2e1f0, 0x8796a5b4, 0x4b5a6978, 0x0f1e2d3c};
	u32 plaintext[LEA_BLK_WORD_LEN] = {0x33323130, 0x37363534, 0x3B3A3938, 0x3F3E3D3C};
	u32 delta[LEA_DELTA_WORD_LEN] = {0xc3efe9db, 0x44626b02, 0x79e27c8a, 0x78df30ec, 0x715ea49e, 0xc785da0a, 0xe04ef22a, 0xe5c40957};
	u32 ciphertext[LEA_BLK_WORD_LEN] = {
		0,
	};

	LEA256_Keyschedule(pdRndKeys, masterKeys);

	LEA256_EncryptBlk(ciphertext, plaintext, pdRndKeys);
	printf("LEA256 ciphertext: 0x%08X, 0x%08X, 0x%08X, 0x%08X\n", ciphertext[0], ciphertext[1], ciphertext[2], ciphertext[3]);

	u32 *u_plain, *u_masterkey, *u_delta, *u_cipher;

	gpuErrorCheck(hipMallocManaged(&u_plain, LEA_BLK_WORD_LEN * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&u_masterkey, LEA256_KEY_WORD_LEN * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&u_delta, LEA_DELTA_WORD_LEN * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&u_cipher, LEA_BLK_WORD_LEN * sizeof(u32)));

	for (int i = 0; i < LEA_BLK_WORD_LEN; i++)
	{
		u_plain[i] = plaintext[i];
	}
	for (int i = 0; i < (LEA256_KEY_WORD_LEN); i++)
	{
		u_masterkey[i] = masterKeys[i];
	}
	for (int i = 0; i < (LEA_DELTA_WORD_LEN); i++)
	{
		u_delta[i] = delta[i];
	}
	for (int i = 0; i < LEA_BLK_WORD_LEN; i++)
	{
		u_cipher[i] = ciphertext[i];
	}

	printf("-------------------------------\n");
	u64 *range = calculateRange();

	clock_t beginTime = clock();
	LEA_256_ES_SharedMemory<<<BLOCKS, THREADS>>>(u_plain, u_cipher, u_masterkey, u_delta, range);
	gpuErrorCheck(hipDeviceSynchronize());
	printf("Time elapsed: %f sec\n", float(clock() - beginTime) / CLOCKS_PER_SEC);
	printf("-------------------------------\n");
	printLastCUDAError();

	hipFree(u_plain);
	hipFree(u_masterkey);
	hipFree(u_delta);
}
