#include "hip/hip_runtime.h"
#include "lea_set.cuh"

void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		//if (abort) exit(code);
	}
}

void printLastCUDAError(){
	hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		printf("-----\n");
		printf("ERROR: hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
		printf("-----\n");
	}
}

__host__ u64* calculateRange() {
	u64* range;
	gpuErrorCheck(hipMallocManaged(&range, 1 * sizeof(u64)));
	int threadCount = BLOCKS * THREADS;
	double keyRange = pow(2, TWO_POWER_RANGE);
	double threadRange = keyRange / threadCount;
	*range = ceil(threadRange);

	printf("Blocks                        : %d\n", BLOCKS);
	printf("Threads                       : %d\n", THREADS);
	printf("Total Thread count            : %d\n", threadCount);
	printf("Key Range (power)             : %d\n", TWO_POWER_RANGE);
	printf("Key Range (decimal)           : %.0f\n", keyRange);
	printf("Each Thread Key Range         : %.2f\n", threadRange);
	printf("Each Thread Key Range (kernel): %llu\n", range[0]);
	printf("Total encryptions             : %.0f\n", ceil(threadRange) * threadCount);
	printf("-------------------------------\n");
	
	return range;
}

void checkDeviceProperties() {
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Max threads dim: %d %d %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Max grid size: %d %d %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Shared memory per block: %lu\n", prop.sharedMemPerBlock);
}

// CUDA kernel function
